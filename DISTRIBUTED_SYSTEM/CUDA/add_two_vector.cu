//#include <cuda_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#define SIZE 1024

/*
void VectorAdd(int *a, int *b, int *c,int n) {
    int i = 0;
    for(; i < n; ++i) {
        c[i] = a[i] + b[i];
    }
}
*/
__global__ void VectorAdd(int *a, int *b, int *c,int n) {
    int i = threadIdx.x;
    
    if( i < n)
        c[i] = a[i] + b[i];
}

int main()
{
    int *a, *b, *c;

    //a = (int *) std::malloc(SIZE * sizeof(int));
    hipMallocManaged(&a, SIZE * sizeof(int));
    //b = (int *) std::malloc(SIZE * sizeof(int));
    hipMallocManaged(&b, SIZE * sizeof(int));
    //c = (int *) std::malloc(SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));

    for(int i = 0; i < SIZE; ++i) {
      a[i] = i;
      b[i] = i;
      c[i] = 0;  
    }  

    //VectorAdd(a, b, c, SIZE);
    VectorAdd  <<<1, SIZE>>> (a, b, c, SIZE); // block and thread size
    hipDeviceSynchronize();

    for(int i = 0; i < 10; ++i) 
        printf("C[%d] = %d \n", i, c[i]);

    //free(a);
    hipFree(a);
    //free(b);
    hipFree(b);
    //free(c);
    hipFree(c);

    return 0;
}